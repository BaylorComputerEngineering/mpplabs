#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__constant__ float M_c[FILTER_SIZE][FILTER_SIZE];

__global__ void convolution(Matrix N, Matrix P)
{
	/********************************************************************
	Determine input and output indexes of each thread
	Load a tile of the input image to shared memory
	Apply the filter on the input image tile
	Write the compute values to the output image at the correct indexes
	********************************************************************/

    //INSERT KERNEL CODE HERE


}
