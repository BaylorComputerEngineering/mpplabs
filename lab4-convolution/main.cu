/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

	Matrix M_h, N_h, P_h; // M: filter, N: input image, P: output image
	Matrix N_d, P_d;
	unsigned imageHeight, imageWidth;
	hipError_t cuda_ret;
	dim3 dim_grid, dim_block;

	/* Read image dimensions */
    if (argc == 1) {
        imageHeight = 600;
        imageWidth = 1000;
    } else if (argc == 2) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[1]);
    } else if (argc == 3) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./convolution          # Image is 600 x 1000"
           "\n    Usage: ./convolution <m>      # Image is m x m"
           "\n    Usage: ./convolution <m> <n>  # Image is m x n"
           "\n");
        exit(0);
    }

	/* Allocate host memory */
	M_h = allocateMatrix(FILTER_SIZE, FILTER_SIZE);
	N_h = allocateMatrix(imageHeight, imageWidth);
	P_h = allocateMatrix(imageHeight, imageWidth);

	/* Initialize filter and images */
	initMatrix(M_h);
	initMatrix(N_h);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Image: %u x %u\n", imageHeight, imageWidth);
    printf("    Mask: %u x %u\n", FILTER_SIZE, FILTER_SIZE);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

	N_d = allocateDeviceMatrix(imageHeight, imageWidth);
	P_d = allocateDeviceMatrix(imageHeight, imageWidth);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

	/* Copy image to device global memory */
	copyToDeviceMatrix(N_d, N_h);

	/* Copy mask to device constant memory */

	cuda_ret = hipMemcpyToSymbol(HIP_SYMBOL(M_c), M_h.elements,
	    M_h.height*M_h.width * sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to copy to constant memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

	dim_block.x = BLOCK_SIZE;
  dim_block.y = BLOCK_SIZE;
  dim_block.z = 1;

	dim_grid.x = imageWidth/TILE_SIZE;
	if(imageWidth%TILE_SIZE != 0) dim_grid.x++;
	dim_grid.y = imageHeight/TILE_SIZE;
	if(imageHeight%TILE_SIZE != 0) dim_grid.y++;
	dim_grid.z = 1;

	convolution<<<dim_grid, dim_block>>>(N_d, P_d);

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    copyFromDeviceMatrix(P_h, P_d);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(M_h, N_h, P_h);

    // Free memory ------------------------------------------------------------

	freeMatrix(M_h);
	freeMatrix(N_h);
	freeMatrix(P_h);
	freeDeviceMatrix(N_d);
	freeDeviceMatrix(P_d);

	return 0;
}
