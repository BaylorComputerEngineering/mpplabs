/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    float* A_d;
    cuda_ret = hipMalloc((void**) &A_d, sizeof(float)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    float* B_d;
    cuda_ret = hipMalloc((void**) &B_d, sizeof(float)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    float* C_d;
    cuda_ret = hipMalloc((void**) &C_d, sizeof(float)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    cuda_ret = hipMemcpy(A_d, A_h, sizeof(float)*n, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(B_d, B_h, sizeof(float)*n, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    const unsigned int THREADS_PER_BLOCK = 512;
    const unsigned int numBlocks = (n - 1)/THREADS_PER_BLOCK + 1;
    dim3 gridDim(numBlocks, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
    vecAddKernel<<< gridDim, blockDim >>> (A_d, B_d, C_d, n);

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    cuda_ret = hipMemcpy(C_h, C_d, sizeof(float)*n, hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory from device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;

}

