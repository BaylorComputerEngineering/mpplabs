
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {

    // Calculate global thread index based on the block and thread indices ----

    //INSERT KERNEL CODE HERE
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use global index to determine which elements to read, add, and write ---

    //INSERT KERNEL CODE HERE
    if (i < n) {
        C[i] = A[i] + B[i];
    }

}

